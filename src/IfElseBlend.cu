
#include <stdio.h>
#include <hip/hip_runtime.h>

#define VECTOR_LENGTH (1<<24)
#define TILE_WIDTH 128		
#define NUM_BLOCKS 180

__global__ void IfElse(const int *control,const long *source1,const long *source2,long *dest, const long vectorLength) 
{	
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int stride = gridDim.x * blockDim.x;
	long idx = bid*blockDim.x + tid;		
	int elementsPerThread = ceil((float)vectorLength/stride);		

	for (int i = 0; i<elementsPerThread; ++i)
		if (idx < vectorLength) {
			if (control[idx])
				dest[idx] = source1[idx];
			else
				dest[idx] = source2[idx];
			idx += stride;
		}
}

// Helper function to handle CUDA errors
void ErrorHandle(hipError_t err) {
	if (err != hipSuccess) {
		printf("Error with CUDA call (error code : %s)\n", hipGetErrorString(err));
		exit(-1);
	}
}

int main(int argc, char **argv)
{
	
	long vectorLength = VECTOR_LENGTH;
	int numBlocks = NUM_BLOCKS;
	int numThreads = TILE_WIDTH;
	
	printf("Number of threads launched is %d\n", numBlocks*numThreads);
	printf("Elements per thread is %li\n", vectorLength/(numBlocks*numThreads));
	
	dim3 dimBlock(numThreads, 1, 1);
	dim3 dimGrid(numBlocks, 1, 1);
	
	// Allocate memory for host side vectors
	long sizeLogic = vectorLength*sizeof(int);
	long sizeData = vectorLength*sizeof(long);
	int * control = (int *)malloc(sizeLogic);
	long * source1 = (long *)malloc(sizeData);
	long * source2 = (long *)malloc(sizeData);
	long * dest = (long *)malloc(sizeData);
	if (control == NULL || source1 == NULL || source2 == NULL || dest == NULL) {
		printf("Unable to allocate host memory\n");
		return(-1);
	}
	
	for (int i = 0; i<vectorLength; ++i) {
		control[i] = i%2;
		source1[i] = 1;
		source2[i] = 2;
	}

	// GPU Vectors
	int * control_d;
	ErrorHandle(hipMalloc((void **)&control_d, sizeLogic));
	ErrorHandle(hipMemcpy(control_d, control, sizeLogic, hipMemcpyHostToDevice));
	
	long * source1_d;
	ErrorHandle(hipMalloc((void **)&source1_d, sizeData));
	ErrorHandle(hipMemcpy(source1_d, source1, sizeData, hipMemcpyHostToDevice));
	
	long * source2_d;
	ErrorHandle(hipMalloc((void **)&source2_d, sizeData));
	ErrorHandle(hipMemcpy(source2_d, source2, sizeData, hipMemcpyHostToDevice));

	long * dest_d;
	ErrorHandle(hipMalloc((void **)&dest_d, sizeData));

	
	
	
	
	
		
	// Kernel Invocation
	IfElse<<<dimGrid, dimBlock>>>(control_d, source1_d, source2_d, dest_d, vectorLength);
	ErrorHandle(hipGetLastError());
	
	// Copy Results Back
	ErrorHandle(hipMemcpy(dest, dest_d, sizeData, hipMemcpyDeviceToHost));

	
	
	
	
	
	
	// Print Control
	printf("Control Vector\n[1]   ");
	for (int i = 0; i < 100; ++i)
		printf("%d  ",control[i]);
	printf("   . . .     ");
	for (int i = vectorLength-100; i < vectorLength; ++i)
		printf("%d  ",control[i]);
	printf("[%li]\n\n",vectorLength);
	
	// Print Results
	printf("Output Vector\n[1]   ");
	for (int i = 0; i < 100; ++i)
		printf("%li  ",dest[i]);
	printf("   . . .     ");
	for (int i = vectorLength-100; i < vectorLength; ++i)
		printf("%li  ",dest[i]);
	printf("[%li]\n",vectorLength);
	
	// Clean up
	ErrorHandle(hipFree(control_d));
	ErrorHandle(hipFree(source1_d));
	ErrorHandle(hipFree(source2_d));
	ErrorHandle(hipFree(dest_d));
	
	free(control);
	free(source1);
	free(source2);
	free(dest);
	
	return 0;
}